#include "hip/hip_runtime.h"
#include "init.h"
#include <uchar.h>

#include <sys/time.h>
#include <stdbool.h>
#include <getopt.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <errno.h>
#include <math.h>
#include <hip/hip_runtime_api.h>

#define TIME_DIFFERENCE(_start, _end) \
    ((_end.tv_sec + _end.tv_nsec / 1.0e9) - \
    (_start.tv_sec + _start.tv_nsec / 1.0e9))

#define TIME_DIFFERENCE_NSEC(_start, _end) \
    ((_end.tv_nsec < _start.tv_nsec)) ? \
    ((_end.tv_sec - 1 - (_start.tv_sec)) * 1e9 + _end.tv_nsec + 1e9 - _start.tv_nsec) : \
    ((_end.tv_sec - (_start.tv_sec)) * 1e9 + _end.tv_nsec - _start.tv_nsec)

#define TIME_DIFFERENCE_GETTIMEOFDAY(_start, _end) \
    ((_end.tv_sec + _end.tv_usec / 1.0e6) - \
    (_start.tv_sec + _start.tv_usec / 1.0e6))


#define TIME_NOW(_t) (clock_gettime(CLOCK_MONOTONIC, (_t)))

#define BUILTIN_CAO

__managed__ hdc_vars hd;

// Number of samples stored in each channel in the dataset
// In the test data sets, these are EMG samples which have been
// sampled at a rate of 500 Hertz
__managed__ int32_t number_of_input_samples;

//__managed__ uint32_t iM[MAX_IM_LENGTH * (MAX_BIT_DIM + 1)];
__managed__ uint32_t *iM;

//__managed__ uint32_t chAM[MAX_CHANNELS * (MAX_BIT_DIM + 1)];
__managed__ __device__ uint32_t *chAM;

/**
 * @brief Exit if NOMEM
 */
void
nomem() {
    fprintf(stderr, "ERROR: No memory\n");
    exit(ENOMEM);
}

/**
 * @brief Read data from @p input_file into globals and test_set
 *
 * @param[in] input_file    File to read from
 * @param[in,out] test_set  Test data to allocate and fill
 */
int
read_data(char const *input_file, double **test_set) {
    int ret = 0;
    errno = 0;
    FILE *file = fopen(input_file, "rb");
    if (file == NULL) {
        return errno != 0 ? errno : -1;
    }

    int32_t version;
    size_t sz = sizeof(version);
    if (fread(&version, 1, sz, file) != sz) {
        return ferror(file);
    }
    if (version != VERSION) {
        fprintf(stderr, "Binary file version (%d) does not match expected (%d)\n", version,
                VERSION);
        return -1;
    }
    if ((fread(&hd.dimension, 1, sz, file) != sz) || (fread(&hd.channels, 1, sz, file) != sz) ||
        (fread(&hd.bit_dim, 1, sz, file) != sz) ||
        (fread(&number_of_input_samples, 1, sz, file) != sz) || (fread(&hd.n, 1, sz, file) != sz) ||
        (fread(&hd.im_length, 1, sz, file) != sz)) {
        return ferror(file);
    }

    sz = hd.channels * number_of_input_samples * sizeof(double);
    //*test_set = (double *)malloc(sz);
    checkCudaErrors(hipMallocManaged(&(*test_set), sz));
    if (*test_set == NULL) {
        nomem();
    }
    if (fread(*test_set, 1, sz, file) != sz) {
        return ferror(file);
    }

    sz = hd.channels * (hd.bit_dim + 1) * sizeof(uint32_t);
    size_t bread;
    bread = fread(chAM, 1, sz, file);

    if (bread != sz) {
        return ferror(file);
    }

    sz = hd.im_length * (hd.bit_dim + 1) * sizeof(uint32_t);
    bread = fread(iM, 1, sz, file);

    if (bread != sz) {
        return ferror(file);
    }

    sz = hd.n * (hd.bit_dim + 1) * sizeof(uint32_t);
    if (fread(hd.aM_32, 1, sz, file) != sz) {
        return ferror(file);
    }

    fclose(file);
    return ret;
}

/**
 * @brief Round a double to an integer.
 *
 * @param[in] num Double to round
 * @return        Rounded integer value
 */
int
round_to_int(double num) {
    return (num - floor(num) > 0.5) ? ceil(num) : floor(num);
}

/**
 * @brief Quantization: each sample is rounded to the nearest integer.
 *
 * @param[out] buffer Rounded integers
 */
void
quantize_set(double const *input_set, int32_t *buffer) {
    for (int i = 0; i < hd.channels; i++) {
        for (int j = 0; j < number_of_input_samples; j++) {
            buffer[(i * number_of_input_samples) + j] =
                round_to_int(input_set[(i * number_of_input_samples) + j]);
        }
    }
}

/**
 * @brief Tests the accuracy based on input testing queries.
 *
 * @param[in] q_32  Query hypervector
 * @param[in] aM_32 Trained associative memory
 * @return          Classification result
 */
__host__ __device__ int
associative_memory_32bit(uint32_t *q_32, uint32_t *aM_32) {
    int sims[CLASSES] = {0};

    // Computes Hamming Distances
    hamming_dist(q_32, aM_32, sims);

    // Classification with Hamming Metric
    return max_dist_hamm(sims);
}

/**
 * @brief Computes the maximum Hamming Distance.
 *
 * @param[in] distances Distances associated to each class
 * @return              The class related to the maximum distance
 */
__host__ __device__ int
max_dist_hamm(int *distances) {
    int max = distances[0];
    int max_index = 0;

    for (int i = 1; i < CLASSES; i++) {
        if (max > distances[i]) {
            max = distances[i];
            max_index = i;
        }
    }

    return max_index;
}

/**
 * @brief Computes the Hamming Distance for each class.
 *
 * @param[in] q     Query hypervector
 * @param[in] aM    Associative Memory matrix
 * @param[out] sims Distances' vector
 */
__host__ __device__ void
hamming_dist(uint32_t *q, uint32_t *aM, int *sims) {
    for (int i = 0; i < CLASSES; i++) {
        sims[i] = 0;
        for (int j = 0; j < hd.bit_dim + 1; j++) {
            sims[i] += number_of_set_bits(q[j] ^ aM[A2D1D(hd.bit_dim + 1, i, j)]);
        }
    }
}

/**
 * @brief Read from im
 * @param[in] im_ind    im array index
 */
__host__ __device__ static inline uint32_t
read_im(uint32_t im_ind) {
    return iM[im_ind];
}

/**
 * @brief Read from cham
 * @param[in] cham_ind    cham array index
 */
__host__ __device__ static inline uint32_t
read_cham(uint32_t cham_ind) {
    return chAM[cham_ind];
}

/**
 * @brief Computes the N-gram.
 *
 * @param[in] input       Input data
 * @param[out] query      Query hypervector
 */
__host__ __device__ void
compute_N_gram(int32_t *input, uint32_t *query) {

    uint32_t chHV[MAX_CHANNELS + 1];

    for (int i = 0; i < hd.bit_dim + 1; i++) {
        query[i] = 0;
        for (int j = 0; j < hd.channels; j++) {
            int ix = input[j];

            uint32_t im = read_im(A2D1D(hd.bit_dim + 1, ix, i));
            uint32_t cham = read_cham(A2D1D(hd.bit_dim + 1, j, i));

            chHV[j] = im ^ cham;
        }
        // this is done to make the dimension of the matrix for the componentwise majority odd.
        chHV[hd.channels] = chHV[0] ^ chHV[1];

        // componentwise majority: compute the number of 1's
        for (int z = 31; z >= 0; z--) {
            uint32_t cnt = 0;
            for (int j = 0; j < hd.channels + 1; j++) {
                uint32_t a = chHV[j] >> z;
                uint32_t mask = a & 1;
                cnt += mask;
            }

            if (cnt > 2) {
                query[i] = query[i] | (1 << z);
            }
        }
    }
}

/**
 * @brief Computes the number of 1's
 *
 * @param i The i-th variable that composes the hypervector
 * @return  Number of 1's in i-th variable of hypervector
 */
__host__ __device__ inline int
number_of_set_bits(uint32_t i) {
#ifdef BUILTIN_CAO
#ifdef  __CUDA_ARCH__
    return __popc(i);
#else
    return __builtin_popcount(i);
#endif
#else
    i = i - ((i >> 1) & 0x55555555);
    i = (i & 0x33333333) + ((i >> 2) & 0x33333333);
    return (((i + (i >> 4)) & 0x0F0F0F0F) * 0x01010101) >> 24;
#endif
}


/**
 * @struct hdc_data
 * @brief HDC data for HDC task
 */
typedef struct hdc_data {
    int32_t *data_set;     /**< Input HDC dataset */
    int32_t *results;      /**< Output from run */
    uint32_t result_len;   /**< Length of the results */
    double execution_time; /**< Total execution time of run */
} hdc_data;

/**
 * @brief Function for @p run_hdc to run HDC task
 */
typedef int (*hdc)(int32_t *data_set, int32_t *results, void *runtime);

/**
 * @brief Run the HDC algorithm for the host
 *
 * @param[in]  data_set  Input dataset
 * @param[out] results   Results from run
 * @param[out] runtime   Runtimes of individual sections (unused)
 *
 * @return               Non-zero on failure.
 */
static int
host_hdc(int32_t *data_set, int32_t *results, void *runtime) {

    (void) runtime;

    uint32_t overflow = 0;
    uint32_t old_overflow = 0;
    uint32_t mask = 1;
    uint32_t q[hd.bit_dim + 1];
    uint32_t q_N[hd.bit_dim + 1];
    int32_t quantized_buffer[hd.channels];

    int result_num = 0;

    for (int ix = 0; ix < number_of_input_samples; ix += hd.n) {

        for (int z = 0; z < hd.n; z++) {

            for (int j = 0; j < hd.channels; j++) {
                if (ix + z < number_of_input_samples) {
                    int ind = A2D1D(number_of_input_samples, j, ix + z);
                    quantized_buffer[j] = data_set[ind];
                }
            }

            // Spatial and Temporal Encoder: computes the n-gram.
            // N.B. if n = 1 we don't have the Temporal Encoder but only the Spatial Encoder.
            if (z == 0) {
                compute_N_gram(quantized_buffer, q);
            } else {
                compute_N_gram(quantized_buffer, q_N);

                // Here the hypervector q is shifted by 1 position as permutation,
                // before performing the componentwise XOR operation with the new query (q_N).
                overflow = q[0] & mask;

                for (int i = 1; i < hd.bit_dim; i++) {
                    old_overflow = overflow;
                    overflow = q[i] & mask;
                    q[i] = (q[i] >> 1) | (old_overflow << (32 - 1));
                    q[i] = q_N[i] ^ q[i];
                }

                old_overflow = overflow;
                overflow = (q[hd.bit_dim] >> 16) & mask;
                q[hd.bit_dim] = (q[hd.bit_dim] >> 1) | (old_overflow << (32 - 1));
                q[hd.bit_dim] = q_N[hd.bit_dim] ^ q[hd.bit_dim];

                q[0] = (q[0] >> 1) | (overflow << (32 - 1));
                q[0] = q_N[0] ^ q[0];
            }
        }
        // classifies the new N-gram through the Associative Memory matrix.
        results[result_num++] = associative_memory_32bit(q, hd.aM_32);
    }

    return 0;
}


__global__ void hdc_kernel(int32_t *data_set, int32_t *results, void *runtime)
{
    uint32_t idx = blockDim.x * blockIdx.x + threadIdx.x;

    if(idx % hd.n == 0) {
        uint32_t overflow = 0;
        uint32_t old_overflow = 0;
        uint32_t mask = 1;
        uint32_t *q = (uint32_t *)malloc(sizeof(uint32_t)*hd.bit_dim + 1);
        uint32_t *q_N = (uint32_t *)malloc(sizeof(uint32_t)*hd.bit_dim + 1);
        int32_t *quantized_buffer = (int32_t *)malloc(sizeof(int32_t)* hd.channels);

        //printf("%d %d %d %d\n",number_of_input_samples, hd.n, hd.channels, hd.bit_dim);
        
        for (int z = 0; z < hd.n; z++) {

            for (int j = 0; j < hd.channels; j++) {
                if (idx + z < number_of_input_samples) {
                    int ind = A2D1D(number_of_input_samples, j, idx + z);
                    quantized_buffer[j] = data_set[ind];
                }
            }

            // Spatial and Temporal Encoder: computes the n-gram.
            // N.B. if n = 1 we don't have the Temporal Encoder but only the Spatial Encoder.
            if (z == 0) {
                compute_N_gram(quantized_buffer, q);
            } else {
                compute_N_gram(quantized_buffer, q_N);

                // Here the hypervector q is shifted by 1 position as permutation,
                // before performing the componentwise XOR operation with the new query (q_N).
                overflow = q[0] & mask;

                for (int i = 1; i < hd.bit_dim; i++) {
                    old_overflow = overflow;
                    overflow = q[i] & mask;
                    q[i] = (q[i] >> 1) | (old_overflow << (32 - 1));
                    q[i] = q_N[i] ^ q[i];
                }

                old_overflow = overflow;
                overflow = (q[hd.bit_dim] >> 16) & mask;
                q[hd.bit_dim] = (q[hd.bit_dim] >> 1) | (old_overflow << (32 - 1));
                q[hd.bit_dim] = q_N[hd.bit_dim] ^ q[hd.bit_dim];

                q[0] = (q[0] >> 1) | (overflow << (32 - 1));
                q[0] = q_N[0] ^ q[0];
            }
        }
        // classifies the new N-gram through the Associative Memory matrix.
        results[idx/hd.n] = associative_memory_32bit(q, hd.aM_32);
    
        free(q);
        free(q_N);
        free(quantized_buffer);
    }
}

static int
gpu_hdc(int32_t *data_set, int32_t *results, void *runtime) {

    //int device = -1;
    //hipGetDevice(&device);
    //hipMemPrefetchAsync(iM, sizeof(uint32_t) * MAX_IM_LENGTH * (MAX_BIT_DIM + 1), device, NULL);
    //hipMemPrefetchAsync(chAM, sizeof(uint32_t) * MAX_CHANNELS * (MAX_BIT_DIM + 1), device, NULL);

    hdc_kernel<<<number_of_input_samples,1>>>(data_set, results, runtime);
    checkCudaErrors(hipDeviceSynchronize());

    return 0;
}

/**
 * @brief Run a HDC workload and time the execution
 *
 * @param[in] fn        Function to run HDC algorithm
 * @param[out] data     Results from HDC run
 * @param[out] runtime  Run times from sections of @p fn
 *
 * @return Non-zero On failure
 */
static double
run_hdc(hdc fn, hdc_data *data, void *runtime) {
    struct timespec start, end;

    int ret = 0;

    uint8_t extra_result = (number_of_input_samples % hd.n) != 0;
    data->result_len = (number_of_input_samples / hd.n) + extra_result;
    uint32_t result_size = data->result_len * sizeof(int32_t);


    //data->results = (int32_t *)malloc(result_size);
    checkCudaErrors(hipMallocManaged(&data->results, result_size));
    
    if (data->results == NULL) {
        nomem();
    }

    TIME_NOW(&start);
    ret = fn(data->data_set, data->results, runtime);
    TIME_NOW(&end);

    data->execution_time = TIME_DIFFERENCE(start, end);

    return ret;
}

/**
 * @brief Compare the results from the host and GPU confirming they are the same
 *        or printing differences
 *
 * @param[in] gpu_data   Results to be tested from GPU
 * @param[in] host_data  Results to be tested from host
 * @param[in] check_only Only check results are equal, dont print differences
 *
 * @return               Non-zero if results are not the same
 */
static int
compare_results(hdc_data *gpu_data, hdc_data *host_data, bool check_only) {
    int ret = 0;

    if (!check_only) {
        printf("--- Compare --\n");
        printf("(%u) results\n", host_data->result_len);
    }

    for (uint32_t i = 0; i < host_data->result_len; i++) {
        if (host_data->results[i] != gpu_data->results[i]) {
            if (check_only) {
                return -1;
            }
            fprintf(stderr, "(host_results[%u] = %d) != (gpu_results[%u] = %d)\n", i,
                    host_data->results[i], i, gpu_data->results[i]);
            ret = -1;
        }
    }

    if (check_only) {
        return 0;
    }

  /*  char *faster;
    double time_diff, percent_diff;
    if (dpu_data->execution_time > host_data->execution_time) {
        faster = "Host";
        time_diff = dpu_data->execution_time - host_data->execution_time;
        percent_diff = dpu_data->execution_time / host_data->execution_time;
    } else {
        faster = "DPU";
        time_diff = host_data->execution_time - dpu_data->execution_time;
        percent_diff = host_data->execution_time / dpu_data->execution_time;
    }

    printf("%s was %fs (%f x) faster\n", faster, time_diff, percent_diff);
*/
    return ret;
}

/**
 * @brief Print results from HDC run
 * @param[in] data  Results to print
 */
static void
print_results(hdc_data *data) {
    for (uint32_t i = 0; i < data->result_len; i++) {
        printf("%d\n", data->results[i]);
    }
}

/**
 * @brief Display usage information to @p stream
 * @param[in] stream    File pointer to write usage to
 * @param[in] exe_name  Name of executable
 */
static void
usage(FILE *stream, char const *exe_name) {
#ifdef DEBUG
    fprintf(stream, "**DEBUG BUILD**\n");
#endif

    fprintf(stream, "usage: %s [ -d ] -i <INPUT_FILE>\n", exe_name);
    fprintf(stream, "\td: use GPU\n");
    fprintf(stream, "\ti: input file\n");
    fprintf(stream, "\tr: show runtime only\n");
    fprintf(stream, "\ts: show results\n");
    fprintf(stream, "\tt: test results\n");
    fprintf(stream, "\th: help message\n");
}

int
main(int argc, char **argv) {
    bool use_gpu = false;
    bool show_results = false;
    bool test_results = false;
    bool runtime_only = false;
    int ret = 0;
    int gpu_ret = 0;
    int host_ret = 0;
    char const options[] = "dsthri:";
    char *input = NULL;

    checkCudaErrors(hipMallocManaged(&iM, sizeof(uint32_t) * MAX_IM_LENGTH * (MAX_BIT_DIM + 1)));
    checkCudaErrors(hipMallocManaged(&chAM, sizeof(uint32_t) * MAX_CHANNELS * (MAX_BIT_DIM + 1)));

    int opt;
    while ((opt = getopt(argc, argv, options)) != -1) {
        switch (opt) {
            case 'd':
                use_gpu = true;
                break;

            case 'i':
                input = optarg;
                break;

            case 's':
                show_results = true;
                break;

            case 't':
                test_results = true;
                break;

            case 'r':
                runtime_only = true;
                break;

            case 'h':
                usage(stdout, argv[0]);
                return EXIT_SUCCESS;

            default:
                usage(stderr, argv[0]);
                return EXIT_FAILURE;
        }
    }

    if (input == NULL) {
        fprintf(stderr, "Please add an input file\n");
        usage(stderr, argv[0]);
        return EXIT_FAILURE;
    }

    double *test_set;
    ret = read_data(input, &test_set);
    if (ret != 0) {
        return ret;
    }

    uint32_t buffer_size = (sizeof(int32_t) * number_of_input_samples * hd.channels);
    //int32_t *data_set = (int32_t *)malloc(buffer_size);
    int32_t *data_set;
    checkCudaErrors(hipMallocManaged(&data_set, buffer_size));

    if (data_set == NULL) {
        nomem();
    }

    quantize_set(test_set, data_set);

    hdc_data gpu_results = {.data_set = data_set, .results = NULL};
    hdc_data host_results = {.data_set = data_set, .results = NULL};

    if (use_gpu || test_results) {
        gpu_ret = run_hdc(gpu_hdc, &gpu_results, NULL);
        if (gpu_ret != 0) {
            goto err;
        }
    }

    if (!use_gpu || test_results) {
        host_ret = run_hdc(host_hdc, &host_results, NULL);
        if (host_ret != 0) {
            goto err;
        }
    }

    if (!use_gpu || test_results) {
        if (!runtime_only) {
            printf("--- Host --\n");
            if (show_results) {
                print_results(&host_results);
            }
            printf("Host took %f\n", host_results.execution_time);
        } else {
            printf("%f\n", host_results.execution_time);
        }
    }

    if (use_gpu || test_results) {
        if (!runtime_only) {
            printf("--- GPU --\n");
            if (show_results) {
                print_results(&gpu_results);
            }
            printf("GPU took %f\n", gpu_results.execution_time);
        } else {
            printf("%f\n", gpu_results.execution_time);
        }
    }

    if (test_results) {
        ret = compare_results(&gpu_results, &host_results, runtime_only);
    }

err:
    //free(data_set);
    checkCudaErrors(hipFree(data_set));
    //free(test_set);
    checkCudaErrors(hipFree(test_set));
    //free(host_results.results);
    checkCudaErrors(hipFree(host_results.results));
    //free(gpu_results.results);
    checkCudaErrors(hipFree(gpu_results.results));

    checkCudaErrors(hipFree(iM));
    checkCudaErrors(hipFree(chAM));

    return (ret + gpu_ret + host_ret);
}
